#include "hip/hip_runtime.h"
#include "utils.h"
#include <stdio.h>
#include "findiff_gpu.h"
#include "findiff.h"

__device__ void calc_iterate(float *unew, float *uold, int n, int m, int idx, int idy, int ind){
	if(1<idy && idy<m){
		if(idx<n){
			unew[ind] = (1.9*uold[ind-2] + 1.5*uold[ind-1] +
                            uold[ind] + 0.5*uold[ind+1] + 0.1*uold[ind+2]);
			unew[ind] /= (float)(5.0);
		}
	}
	__syncthreads();
}

__device__ void glob_shared_cpy(float *u_glob, float *unew, float *uold, int pitch, int n, int m, int idx, int idy, int ind){
	// READING DATA FROM GLOBAL MEMORY TO SHARED //
    if(idy<m && idx<n){
		if(threadIdx.y==0 && 0<blockIdx.y){
			unew[ind-2] = u_glob[(idy-2) + idx*pitch];
			unew[ind-1] = u_glob[(idy-1) + idx*pitch];	
			uold[ind-2] = unew[ind-2];
			uold[ind-1] = unew[ind-1];
		}

		unew[ind] = u_glob[idy+idx*pitch];
		uold[ind] = unew[ind];
			
		if(threadIdx.y==(blockDim.y-1) || idy==(m-1)){
			unew[ind+1] = u_glob[(idy+1)%m + idx*pitch];
			unew[ind+2] = u_glob[(idy+2)%m + idx*pitch];
            uold[ind+1] = unew[ind+1];
            uold[ind+2] = unew[ind+2];
        }
    }
}

__device__ void shared_glob_cpy(float *u_glob, float *unew, int pitch, int n, int m, int idx, int idy, int ind){
    if(1<idy && idy<m)
		if(idx<n)
		    u_glob[idy+idx*pitch] = unew[ind];
}

__global__ void iterate_gpu(float *u_glob, int pitch, int n, int m){
	int idx = blockIdx.x*blockDim.x + threadIdx.x;
	int idy = blockIdx.y*blockDim.y + threadIdx.y;
	int ind = threadIdx.y + 2;
    float *uold, *unew;
    extern __shared__ float s[];
    
    unew = &(s[0]);
    uold = &(s[blockDim.y+4]);
    
	// initialising shared memory //
    glob_shared_cpy(u_glob, unew, uold, pitch, n, m, idx, idy, ind);     
    
    // iterating and updating unew //
	calc_iterate(unew, uold, n, m, idx, idy, ind);
    
    // sending vals back to global mem //
    shared_glob_cpy(u_glob, unew, pitch, n, m, idx, idy, ind);
}

__global__ void iterate_gpu_slow(float* unew_glob, float* uold_glob, int n, int m){
	int idx = blockIdx.x*blockDim.x + threadIdx.x;
	int idy = blockIdx.y*blockDim.y + threadIdx.y;
    if(1<idy && idy<m){
		if(idx<n){
			unew_glob[idy+idx*m] = (1.9*uold_glob[(idy+idx*m)-2] + 1.5*uold_glob[(idy+idx*m)-1] +
                        uold_glob[idy+idx*m] + 0.5*uold_glob[(idy+1)%m+idx*m] 
                            + 0.1*uold_glob[(idy+2)%m+idx*m]);
			unew_glob[idy+idx*m] /= (float)(5.0);
		}
	}
}

__global__ void red_rows(float* u_glob, float* u_glob_out, int pitch, int n, int m){
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    int idy = blockIdx.y*blockDim.y + threadIdx.y;
    int ind = threadIdx.y;
    extern __shared__ float tmp[];
    int i, disp;

    if(idy<m && idx<n)
        tmp[ind] = u_glob[idy+idx*pitch];
    
    disp = (1+blockIdx.y)*blockDim.y;
    i = (disp > m) ? (blockDim.y - (disp-m)):blockDim.y;
    /*
    if(idx==0 && idy ==0){
        printf("shared[0,1,2] = %f, %f, %f, i=%d\n",tmp[0], tmp[1], tmp[2], i);
        printf("m = %d, disp = %d, m mod disp = %d\n", m, disp, 6%3);
    }
    */
    for( ; i>1; i>>=1){
        if(ind<(i/2)){
            tmp[ind] += tmp[ind+(i/2)];
            if(ind==0 && i%2!=0)
                tmp[ind] += tmp[ind+i-1]; 
        }
         __syncthreads();
    }
    if(ind==0)
        u_glob_out[blockIdx.y + idx*pitch] = tmp[0];
}

__global__ void red_rows_glob(float* u_glob, float* u_glob_out, int pitch, int n, int m){
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    int idy = blockIdx.y*blockDim.y + threadIdx.y;

    if(idy < (m/2) && idx < n){
        u_glob_out[idy + idx*pitch] += u_glob[idy + (m/2) + idx*pitch];
        if(m%2!=0 && idy==0)
            u_glob_out[idy + idx*pitch] += u_glob[idy + (m-1) + idx*pitch];
    }
}

extern "C" {
void fdiff_gpu(float *u_vals, float *temps, int n, int m, int p, int block_size_Y, Tau* tau, int mallocPitch, int red){
    float *u_glob;
    size_t u_glob_size;
    int i, pitch, m_tmp;
	hipEvent_t start, finish;

    hipEventCreate(&start);
    hipEventCreate(&finish);
    
    hipEventRecord(start, 0);
    if(!mallocPitch){
        hipMalloc( (void**)&u_glob, n*m*sizeof(float));
        hipEventRecord(finish, 0);
        hipEventSynchronize(finish);
        hipEventElapsedTime(&tau->alloc_GPU, start, finish);
        
        hipEventRecord(start,0);
        hipMemcpy(u_glob, u_vals, n*m*sizeof(float), hipMemcpyHostToDevice);
        hipEventRecord(finish, 0);
        hipEventSynchronize(finish);
        hipEventElapsedTime(&tau->transf_GPU, start, finish);
        
        pitch = m;
    } else {
        hipMallocPitch( (void**)&u_glob, &u_glob_size, (size_t)(m*sizeof(float)), n);
        hipEventRecord(finish, 0);
        hipEventSynchronize(finish);
        hipEventElapsedTime(&tau->alloc_GPU, start, finish);
        
        hipEventRecord(start,0);
        hipMemcpy2D(u_glob, u_glob_size, u_vals, m*sizeof(float), m*sizeof(float), n, hipMemcpyHostToDevice);
        hipEventRecord(finish, 0);
        hipEventSynchronize(finish);
        hipEventElapsedTime(&tau->transf_GPU, start, finish);
        
        pitch = (int)u_glob_size/sizeof(float);
    }

    dim3 dimBlock(1, block_size_Y);
    dim3 dimGrid((n/dimBlock.x)+(!(n%dimBlock.x)?0:1), (m/dimBlock.y)+(!(m%dimBlock.y)?0:1));
   
    hipEventRecord(start, 0);
    for(i=0;i<p;i++)
	    iterate_gpu<<<dimGrid,dimBlock,2*(block_size_Y+4)*sizeof(float)>>>(u_glob, pitch, n, m);
    
    hipEventRecord(finish, 0);
    hipEventSynchronize(finish);
    hipEventElapsedTime(&tau->calc_GPU, start, finish);
    
    hipEventRecord(start, 0);
    if(!mallocPitch)
        hipMemcpy(u_vals, u_glob, n*m*sizeof(float), hipMemcpyDeviceToHost);
    else
        hipMemcpy2D(u_vals, m*sizeof(float), u_glob, u_glob_size, m*sizeof(float), n, hipMemcpyDeviceToHost);
    hipEventRecord(finish, 0);
    hipEventSynchronize(finish);
    hipEventElapsedTime(&tau->transf_RAM, start, finish);
    
    if(red){  
        hipEventRecord(start, 0);
        m_tmp = m;
        while(m_tmp > 1){
            printf("m_tmp = %d\n", m_tmp);
            red_rows<<<dimGrid,dimBlock,dimBlock.y*sizeof(float)>>>(u_glob, u_glob, pitch, n, m_tmp);
            m_tmp = (m_tmp/dimBlock.y)+(!(m_tmp%dimBlock.y)?0:1);
        }
        hipEventRecord(finish, 0);
        hipEventSynchronize(finish);
        hipEventElapsedTime(&tau->calc_avg, start, finish);
        
        if(!mallocPitch){
            for(i=0;i<n;i++)
                hipMemcpy(&temps[i], &u_glob[i*m], sizeof(float), hipMemcpyDeviceToHost);
        } else {
            hipMemcpy2D(temps, sizeof(float), &u_glob[0], u_glob_size, sizeof(float), n, hipMemcpyDeviceToHost);
        }
    }

    hipFree(u_glob);
}

void fdiff_gpu_glob(float* u_vals, float* temps, int n, int m, int p, int block_size, Tau* tau, int red){
	float *uold_glob, *unew_glob, *tmp;
    int i, m_tmp;
	hipEvent_t start, finish;

    hipEventCreate(&start);
    hipEventCreate(&finish);
    
    hipEventRecord(start, 0);
    hipMalloc( (void**)&unew_glob, n*m*sizeof(float));
    hipMalloc( (void**)&uold_glob, n*m*sizeof(float));
    hipEventRecord(finish, 0);
    hipEventSynchronize(finish);
    hipEventElapsedTime(&tau->alloc_GPU, start, finish);

    hipEventRecord(start, 0);
    hipMemcpy(unew_glob, u_vals, n*m*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(uold_glob, u_vals, n*m*sizeof(float), hipMemcpyHostToDevice);
    hipEventRecord(finish, 0);
    hipEventSynchronize(finish);
    hipEventElapsedTime(&tau->transf_GPU, start, finish);
    
    dim3 dimBlock(block_size, block_size);
    dim3 dimGrid ((n/dimBlock.x)+(!(n%dimBlock.x)?0:1), (m/dimBlock.y)+(!(m%dimBlock.y)?0:1));

    hipEventRecord(start, 0);
    for(i=0;i<p;i++){
        if(i%2==0)
	        iterate_gpu_slow<<<dimGrid,dimBlock>>>(unew_glob, uold_glob, n, m);
        else
	        iterate_gpu_slow<<<dimGrid,dimBlock>>>(uold_glob, unew_glob, n, m);
    }
    hipEventRecord(finish, 0);
    hipEventSynchronize(finish);
    hipEventElapsedTime(&tau->calc_GPU, start, finish);
    
    hipEventRecord(start, 0);
    if(p%2==0)
        tmp = uold_glob;
    else
        tmp = unew_glob;
    hipMemcpy(u_vals, tmp, n*m*sizeof(float), hipMemcpyDeviceToHost);
    hipEventRecord(finish, 0);
    hipEventSynchronize(finish);
    hipEventElapsedTime(&tau->transf_RAM, start, finish);
    
    m_tmp = m;
    if(red){    
        hipEventRecord(start, 0);
        for( ; m_tmp>1; m_tmp>>=1)
            red_rows_glob<<<dimGrid,dimBlock>>>(tmp, tmp, m, n, m_tmp);
        hipEventRecord(finish, 0);
        hipEventSynchronize(finish);
        hipEventElapsedTime(&tau->calc_avg, start, finish);
        for(i=0;i<n;i++)
            hipMemcpy(&temps[i], &tmp[i*m], sizeof(float), hipMemcpyDeviceToHost);
    }
    
    hipFree(unew_glob); hipFree(uold_glob);
}
}
