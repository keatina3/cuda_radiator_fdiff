#include "hip/hip_runtime.h"
#include "utils.h"
#include "findiff_gpu.h"

__device__ void calc_iterate(float *unew, float *uold, int n, int m, int idx, int idy, int ind){
	if(1<idy && idy<m){
		if(idx<n){
			unew[ind] = (1.9*uold[ind-2] + 1.5*uold[ind-1] +
                            uold[ind] + 0.5*uold[ind+1] + 0.1*uold[ind+2]);
			unew[ind] /= (float)(5.0);
		}
	}
	__syncthreads();
}

__device__ void glob_shared_cpy(float *u_glob, float *unew, float *uold, int pitch, int n, int m, int idx, int idy, int ind){
	// READING DATA FROM GLOBAL MEMORY TO SHARED //
    if(idy<m && idx<n){
		if(threadIdx.y==0 && 0<blockIdx.y){
			unew[ind-2] = u_glob[(idy-2) + idx*pitch];
			unew[ind-1] = u_glob[(idy-1) + idx*pitch];	
			uold[ind-2] = unew[ind-2];
			uold[ind-1] = unew[ind-1];
		}

		unew[ind] = u_glob[idy+idx*pitch];
		uold[ind] = unew[ind];
			
		if(threadIdx.y==(blockDim.y-1) || idy==(m-1)){
			unew[ind+1] = u_glob[(idy+1)%m + idx*pitch];
			unew[ind+2] = u_glob[(idy+2)%m + idx*pitch];
            uold[ind+1] = unew[ind+1];
            uold[ind+2] = unew[ind+2];
        }
    }
}

__device__ void shared_glob_cpy(float *u_glob, float *unew, int pitch, int n, int m, int idx, int idy, int ind){
    if(1<idy && idy<m)
		if(idx<n)
		    u_glob[idy+idx*pitch] = unew[ind];
}

__global__ void iterate_gpu(float *u_glob, int pitch, int n, int m){
	int idx = blockIdx.x*blockDim.x + threadIdx.x;
	int idy = blockIdx.y*blockDim.y + threadIdx.y;
	int ind = threadIdx.y + 2;
    float *uold, *unew;
    extern __shared__ float s[];
    
    unew = &(s[0]);
    uold = &(s[blockDim.y+4]);
    
	// initialising shared memory //
    glob_shared_cpy(u_glob, unew, uold, pitch, n, m, idx, idy, ind);     
    
    // iterating and updating unew //
	calc_iterate(unew, uold, n, m, idx, idy, ind);
    
    // sending vals back to global mem //
    shared_glob_cpy(u_glob, unew, pitch, n, m, idx, idy, ind);
}

__global__ void iterate_gpu_slow(float* unew_glob, float* uold_glob, int n, int m){
	int idx = blockIdx.x*blockDim.x + threadIdx.x;
	int idy = blockIdx.y*blockDim.y + threadIdx.y;
    if(1<idy && idy<m){
		if(idx<n){
			unew_glob[idy+idx*m] = (1.9*uold_glob[(idy+idx*m)-2] + 1.5*uold_glob[(idy+idx*m)-1] +
                        uold_glob[idy+idx*m] + 0.5*uold_glob[(idy+1)%m+idx*m] 
                            + 0.1*uold_glob[(idy+2)%m+idx*m]);
			unew_glob[idy+idx*m] /= (float)(5.0);
		}
	}
}

__global__ void red_rows(float* u_glob, float* u_glob_out, int pitch, int n, int m){
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    int idy = blockIdx.y*blockDim.y + threadIdx.y;
    int ind = threadIdx.y;
    extern __shared__ float tmp[];
    int i, disp;

    if(idy<m && idx<n)
        tmp[ind] = u_glob[idy+idx*pitch];         

    disp = (1+blockIdx.y)*blockDim.y;
    i = disp > m ? (blockDim.y - disp%m):blockDim.y;
    for( ; i>1; i>>=1){
        if(ind<(i/2)){
            tmp[ind] += tmp[ind+(i/2)];
            if(ind==0 && i%2!=0)
                tmp[ind] += tmp[ind+i-1]; 
        }
        __syncthreads();
    }

    if(ind==0)
        u_glob_out[blockIdx.y + idx*pitch] = tmp[0];
}

extern "C" {
void fdiff_gpu(float *u_vals, float *temps, int n, int m, int p, int block_size_Y, Tau* tau){
    float *u_glob;
    size_t u_glob_size;
    int i, pitch, m_tmp;
	
    //hipMalloc( (void**)&u_glob, n*m*sizeof(float));
    //hipMemcpy(u_glob, u_vals, n*m*sizeof(float), hipMemcpyHostToDevice);
    hipMallocPitch( (void**)&u_glob, &u_glob_size, (size_t)(m*sizeof(float)), n);
    hipMemcpy2D(u_glob, u_glob_size, u_vals, m*sizeof(float), m*sizeof(float), n, hipMemcpyHostToDevice);
    pitch = (int)u_glob_size/sizeof(float);
    // pitch = m;

    dim3 dimBlock(1, block_size_Y);
    dim3 dimGrid((n/dimBlock.x)+(!(n%dimBlock.x)?0:1), (m/dimBlock.y)+(!(m%dimBlock.y)?0:1));
   
    for(i=0;i<p;i++)
	    iterate_gpu<<<dimGrid,dimBlock,2*(block_size_Y+4)*sizeof(float)>>>(u_glob, pitch, n, m);
    
    //hipMemcpy(u_vals, u_glob, n*m*sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy2D(u_vals, m*sizeof(float), u_glob, u_glob_size, m*sizeof(float), n, hipMemcpyDeviceToHost);
    
    m_tmp = m;
    while(m_tmp > 1){
        red_rows<<<dimGrid,dimBlock,block_size_Y>>>(u_glob, u_glob, pitch, n, m_tmp);
        m_tmp = (m_tmp/dimBlock.y)+(!(m_tmp%dimBlock.y)?0:1);
    }
    hipMemcpy2D(temps, sizeof(float), u_glob, u_glob_size, sizeof(float), n, hipMemcpyDeviceToHost);

    hipFree(u_glob);
}

void fdiff_gpu_slow(float* u_vals, int n, int m, int p, int block_size, Tau* tau){
	float *uold_glob, *unew_glob;
    int i;
    
    hipMalloc( (void**)&unew_glob, n*m*sizeof(float));
    hipMalloc( (void**)&uold_glob, n*m*sizeof(float));

    hipMemcpy(unew_glob, u_vals, n*m*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(uold_glob, u_vals, n*m*sizeof(float), hipMemcpyHostToDevice);
    
    dim3 dimBlock(block_size, block_size);
    dim3 dimGrid ((n/dimBlock.x)+(!(n%dimBlock.x)?0:1), (m/dimBlock.y)+(!(m%dimBlock.y)?0:1));

    for(i=0;i<p;i++){
        if(i%2==0)
	        iterate_gpu_slow<<<dimGrid,dimBlock>>>(unew_glob, uold_glob, n, m);
        else
	        iterate_gpu_slow<<<dimGrid,dimBlock>>>(uold_glob, unew_glob, n, m);
    }
    
    if(p%2==0)
        hipMemcpy(u_vals, unew_glob, n*m*sizeof(float), hipMemcpyDeviceToHost);
    else
        hipMemcpy(u_vals, uold_glob, n*m*sizeof(float), hipMemcpyDeviceToHost);
	
    hipFree(unew_glob); hipFree(uold_glob);
}
}
